#include "hip/hip_runtime.h"
/*
 * Author:  Jens Breitbart, http://www.gpuified.de/contact/
 *
 * Copyright: See COPYING file that comes with this distribution
 *
 */
// includes, system
#include <cstdlib>
#include <cstdio>
#include <cstring>

#include "memory1d.h"

__global__ void global_function (cupp::memory1d<int> p) {
	#if defined (WE_WANT_OPENMP)
		Loop over all grid cells
			Loop over all blocks in parallel

			To get it working we would just need to:
			- rewrite the kernel operator()
			- replace __synchthreads() with a omp barrier
			- feed this function with the grid/block size
			- and cope with millions of differentes between CUDA and the OpenMP Standard (or let the developer deal with it)
	#endif
	p[threadIdx.x]*=2;
}

typedef void(*kernelT)(cupp::memory1d<int>);

kernelT get_kernel() {
	return global_function;
}
