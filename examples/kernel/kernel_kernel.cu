#include "hip/hip_runtime.h"
/*
 * Author:  Jens Breitbart, http://www.gpuified.de/contact/
 *
 * Copyright: See COPYING file that comes with this distribution
 *
 */

#include "kernel_t.h"

__global__ void global_function (const int i, int &j) {
	j = i;
}

kernelT get_kernel() {
	return global_function;
}
