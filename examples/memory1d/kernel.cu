#include "hip/hip_runtime.h"
/*
 * Author:  Jens Breitbart, http://www.gpuified.de/contact/
 *
 * Copyright: See COPYING file that comes with this distribution
 *
 */
// includes, system
#include <cstdlib>
#include <cstdio>
#include <cstring>

#include "memory1d.h"

__global__ void real_GPU (int* p) {
	p[threadIdx.x]*=2;
}

void kernel(cupp::memory1d<int> &p) {
	using namespace cupp;
	// set up the enviroment
	dim3 block_dim (8);
	dim3 grid_dim  (1);
	
	// start the kernel
	real_GPU <<< grid_dim, block_dim >>> (p.cuda_pointer());

	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		fprintf( stderr, "Cuda error: KERNEL EXEC FAILED in file '%s' in line %i : %s.\n",
		__FILE__, __LINE__, hipGetErrorString( err) );
		exit(EXIT_FAILURE);
	}

}
