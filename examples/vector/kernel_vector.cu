#include "hip/hip_runtime.h"
/*
 * Author:  Jens Breitbart, http://www.gpuified.de/contact/
 *
 * Copyright: See COPYING file that comes with this distribution
 *
 */

#include "deviceT/vector.h"
#include "kernel_t.h"

__global__ void global_function (cupp::deviceT::vector<int> &i) {
	i[threadIdx.x] *= 2;
}

kernelT get_kernel() {
	return global_function;
}
