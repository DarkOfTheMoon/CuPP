#include "hip/hip_runtime.h"
/*
 * Author:  Jens Breitbart, http://www.gpuified.de/contact/
 *
 * Copyright: See COPYING file that comes with this distribution
 *
 */

#include "deviceT/vector.h"
#include "kernel_t.h"

using namespace cupp;

__global__ void global_function (deviceT::vector< deviceT::vector <int> > &i) {
	i[threadIdx.x][0] *= 2;
}

kernelT get_kernel() {
	return global_function;
}
